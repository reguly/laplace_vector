/* Copyright (c) 2012, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <math.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <hip/hip_fp16.h>

__global__ void stencil(int imax, int jmax, __half *Anew, __half *A) {
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;
  if (i < imax+1 && j < jmax+1 && i>0 && j>0) {
    __half val = __hadd(__hadd(A[(j)*(imax+2)+i+1], A[(j)*(imax+2)+i-1]),
                       __hadd(A[(j-1)*(imax+2)+i], A[(j+1)*(imax+2)+i]));
    Anew[(j)*(imax+2)+i] = __hmul(val, __float2half(0.25f));
  }
}

__global__ void copy(int imax, int jmax, __half *Anew, __half *A) {
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;
  if (i < imax+1 && j < jmax+1 && i>0 && j>0)
    A[(j)*(imax+2)+i] = Anew[(j)*(imax+2)+i];
}

int main(int argc, char** argv)
{
    //Size along y
    int jmax = 8190;
    //Size along x
    int imax = 8190;
    int iter_max = 2;

    const double pi  = 2.0 * asin(1.0);
    const __half tol = __float2half(1.0e-3f); // Adjusted tolerance for half precision
    __half error     = __float2half(1.0f);

    __half *A;
    __half *Anew;
    float *y0; // Keep y0 as float for boundary condition calculations

    A    = (__half *)malloc((imax+2) * (jmax+2) * sizeof(__half));
    Anew = (__half *)malloc((imax+2) * (jmax+2) * sizeof(__half));
    y0   = (float *)malloc((imax+2) * sizeof(float));

    // Initialize arrays with zeros
    for (int i = 0; i < (imax+2) * (jmax+2); i++) {
        A[i] = __float2half(0.0f);
    }

    // set boundary conditions
    for (int i = 0; i < imax+2; i++)
      A[(0)*(imax+2)+i]   = __float2half(0.0f);

    for (int i = 0; i < imax+2; i++)
      A[(jmax+1)*(imax+2)+i] = __float2half(0.0f);

    for (int j = 0; j < jmax+2; j++)
    {
        y0[j] = sin(pi * j / (jmax+1));
        A[(j)*(imax+2)+0] = __float2half(y0[j]);
    }

    for (int j = 0; j < imax+2; j++)
    {
        y0[j] = sin(pi * j/ (jmax+1));
        A[(j)*(imax+2)+imax+1] = __float2half(y0[j]*exp(-pi));
    }

    printf("Jacobi relaxation Calculation (Half Precision): %d x %d mesh\n", imax+2, jmax+2);


    
    int iter = 0;

    for (int i = 1; i < imax+2; i++)
       Anew[(0)*(imax+2)+i]   = __float2half(0.0f);

    for (int i = 1; i < imax+2; i++)
       Anew[(jmax+1)*(imax+2)+i] = __float2half(0.0f);

    for (int j = 1; j < jmax+2; j++)
        Anew[(j)*(imax+2)+0]   = __float2half(y0[j]);

    for (int j = 1; j < jmax+2; j++)
        Anew[(j)*(imax+2)+jmax+1] = __float2half(y0[j]*expf(-pi));

    __half *d_A, *d_Anew;
    size_t size = (imax+2)*(jmax+2)*sizeof(__half);
    hipMalloc(&d_A, size);
    hipMalloc(&d_Anew, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_Anew, Anew, size, hipMemcpyHostToDevice);
    
    // CUDA timing variables
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    while ( __hge(error, tol) && iter < iter_max )
    {
        error = __float2half(1.0f);
        dim3 threads(32,8);
        dim3 blocks((imax+2)/32, (jmax+2)/8);
        stencil<<<blocks, threads>>>(imax, jmax, d_Anew, d_A);

        copy<<<blocks, threads>>>(imax, jmax, d_Anew, d_A);
        if(iter % 100 == 0) printf("%5d, %0.6f\n", iter, __half2float(error));

        iter++;
    }

    // Stop timing and calculate runtime
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float runtime;
    hipEventElapsedTime(&runtime, start, stop);
    runtime /= 1000.0f; // Convert from milliseconds to seconds

    printf(" total: %f s\n", runtime);
    
    // Clean up CUDA events and memory
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_Anew);
    free(A);
    free(Anew);
    free(y0);
} 